#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
#include <>

int main(void) {
	//struct containing info such as name, threads/block, etc.
	hipDeviceProp_t devProp;
	int count;
	//pass addr of var, get method populates 
	hipGetDeviceCount(&count);
	for (int i = 0; i < count; i++)
	{
		hipGetDeviceProperties(&devProp, i);
		cout << "Name: " << devProp.name << endl;
		cout << "Clock rate: " << devProp.clockRate << endl;
		cout << "Total global memory: " << devProp.totalGlobalMem << endl;
		printf("Max thread dimensions: (%d, %d, %d)\n", devProp.maxThreadsDim[0], devProp.maxThreadsDim[1], devProp.maxThreadsDim[2]);
	}
}