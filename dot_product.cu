#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include<iostream>
#include "cpu_bitmap.h"
using namespace std;
#include <>

/*
CUDA C has a __shared__ memory section where a copy of var is made for each block & threads within a block can all access var but cannot see or modify copy in other blocks
Must synchronize, if thread A writes to var and thread B wants to modify, must wait and ensure write is done else race condition occurs where correctness of var unknown
Addl shared mem buffers are physically on GPU, as opposed to off-chip DRAM which makes for much faster calls and reduced latency 
*/

