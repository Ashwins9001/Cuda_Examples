#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

#ifndef __CUDACC__ 
#define __CUDACC__
#endif
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>



/*
CUDA C has a __shared__ memory section where a copy of var is made for each block & threads within a block can all access var but cannot see or modify copy in other blocks
Must synchronize, if thread A writes to var and thread B wants to modify, must wait and ensure write is done else race condition occurs where correctness of var unknown
Addl shared mem buffers are physically on GPU, as opposed to off-chip DRAM which makes for much faster calls and reduced latency 
*/

#define imin(a, b) (a < b ? a : b)

const int N = 33 * 1024;
const int threadsPerBlock = 256;

__global__ void dot(float *a, float *b, float *c)
{
	__shared__ float cache[threadsPerBlock]; //arr of caches equal to size of 256, each thread has spot to store temp vals & must wait for all writes to finish before another iteration where val retrieved for more modification 
	int threadID = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheID = threadIdx.x;
	float temp = 0;
	//iteratively take sum of products (dot prod) by creating threads for each arr elem
	//all threads will technically run at same hardware location, however abstract into larger set incase vector exceeds length
	//for small enough vals each thread theoretically computes single sum, yet again for large vecs, can continue greater iterations
	while (threadID < N)
	{
		temp += a[threadID] * b[threadID];
		threadID += blockDim.x * gridDim.x;
	}
	cache[cacheID] = temp; //shared mem buffer to store running sum per thread 
	//Apply reduction to sum vals, whereby input arr made into smaller output arr
	//Apply multiple threads for sum, each one adds two vals of cache[], resulting in log2(threadsPerBlock) steps 
	//Each thread does two computations, therefore 2x per thread. There are threadsPerBlock running in parallel, thus 2^threadsPerBlock computations being done per step
	int i = blockDim.x / 2; //each thread does two tasks, thus need half as many
	while (i != 0) //run in parallel so each step halves size of cache until reach 1 elem in arr
	{
		if (cacheID < i) //check cacheID being summed less than num operators
		{
			cache[cacheID] += cache[cacheID + i]; //add curr cache val to ith 
		}
		__syncthreads();
		i /= 2; //every other cache index
	}
	//Final reduction, each block has single sum left & store to global mem
	if (cacheID == 0)
	{
		c[blockIdx.x] = cache[0]; //send to curr block 
	}
}

int main(void) {}