#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

#define N 5

__global__ void add(int* a, int* b, int* c)
{
	int id = threadIdx.x;
	if (id < N)
	{
		c[id] = b[id] + a[id];
	}
}

int main(void) {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	hipMalloc((void**)&dev_a, sizeof(int) * N);
	hipMalloc((void**)&dev_b, sizeof(int) * N);
	hipMalloc((void**)&dev_c, sizeof(int) * N);
	for (int i = 0; i < N; i++)
	{
		a[i] = -i;
		b[i] = i * i;
	}
	hipMemcpy(dev_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, sizeof(int) * N, hipMemcpyHostToDevice);
	//Invoke device, N within <<<>>> defines N threads are allocated to run in parallel within one block 
	add <<<1, N>>> (dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++)
	{
		cout << "Element at " << i << " is: " << c[i] << endl;
	}
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}


